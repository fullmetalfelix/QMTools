#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <assert.h>
#include "cube.h"
#include "convolve.h"
#include "vectors.h"



/// Number of cubes.
__constant__ unsigned int c_nCubes;

/// Total number of points in a cube grid.
__constant__ unsigned int c_cubeNpts;


/// Number of points along one side of the cube.
__constant__ int c_cubeSide;


/// Total number of GPU blocks assigned to one cube.
__constant__ int c_blocksPerCube;

/// Number of GPU blocks long one side of the cube.
__constant__ int c_blocksPerSide;


/// Diffusion constant of the A0 field.
__constant__ number c_A0_diff;

/// A0 generation factor.
__constant__ number c_A0_gen;

/// A0 spatial loss factor
__constant__ number c_A0_loss[3];



/* FLEXIBLE CUBE SIZE IMPLEMENTATION */

__constant__ uint3 c_cubeSideF;
__constant__ uint3 c_blocksPerCubeF;
__constant__ uint3 c_blocksPerSideF;








/// Loads a reference cube from a file into the CPU memory.
///
///
void cube_load_reference(Cube *cube, const char *filename) {

	printf("loading reference cube: %s ...\n", filename);

	FILE *fbin = fopen(filename, "rb");

	// read the molecule specification
	int natm;
	fread(&natm, sizeof(int), 1, fbin);

	cube->molecule.natoms = natm;
	cube->molecule.Zs = (int*)malloc(sizeof(int) * natm);
	cube->molecule.coords = (float3*)malloc(sizeof(float3) * natm);
	cube->molecule.qtot = 0;

	for (int i=0; i<natm; ++i) {
		fread(&cube->molecule.Zs[i], sizeof(int), 1, fbin);
		fread(&cube->molecule.coords[i], sizeof(float3), 1, fbin);
		cube->molecule.qtot += cube->molecule.Zs[i];
	}
	
	// read the grid shape
	fread(&cube->gridSize.x, sizeof(unsigned int), 1, fbin);
	fread(&cube->gridSize.y, sizeof(unsigned int), 1, fbin);
	fread(&cube->gridSize.z, sizeof(unsigned int), 1, fbin);
	cube->npts = cube->gridSize.x * cube->gridSize.y * cube->gridSize.z;
	printf("grid shape: [%i %i %i]\n", cube->gridSize.x, cube->gridSize.y, cube->gridSize.z);

	cube->maxside = cube->gridSize.x;
	if(cube->gridSize.y > cube->maxside) cube->maxside = cube->gridSize.y;
	if(cube->gridSize.z > cube->maxside) cube->maxside = cube->gridSize.z;


	cube->gpu_grid.x = cube->gridSize.x / 8;
	cube->gpu_grid.y = cube->gridSize.y / 8;
	cube->gpu_grid.z = cube->gridSize.z / 8;
	printf("GPU GRID: [%i %i %i]\n", cube->gpu_grid.x, cube->gpu_grid.y, cube->gpu_grid.z);


	// read the grid origin
	fread(&cube->grid0.x, sizeof(float), 1, fbin);
	fread(&cube->grid0.y, sizeof(float), 1, fbin);
	fread(&cube->grid0.z, sizeof(float), 1, fbin);
	printf("grid origin: %f %f %f \n", cube->grid0.x, cube->grid0.y, cube->grid0.z);

	// translate coordinates so that 0 is at the origin of the grid
	for (int i=0; i<natm; ++i) {
		cube->molecule.coords[i].x -= cube->grid0.x;
		cube->molecule.coords[i].y -= cube->grid0.y;
		cube->molecule.coords[i].z -= cube->grid0.z;
	}

	// read the density cube
	cube->Q = (number*)malloc(sizeof(number) * cube->npts);
	fread(cube->Q, sizeof(number), cube->npts, fbin);

	fclose(fbin);
	printf("cube read.\n");
}

void cube_load_reference_dummy(Cube *cube) {

	printf("loading reference dummy...\n");

	// read the molecule specification
	int natm = 1;
	cube->molecule.natoms = natm;
	cube->molecule.Zs = (int*)malloc(sizeof(int) * natm);
	cube->molecule.coords = (float3*)malloc(sizeof(float3) * natm);

	cube->molecule.Zs[0] = 2;
	cube->molecule.coords[0].x = 0;
	cube->molecule.coords[0].y = 0;
	cube->molecule.coords[0].z = 0;
	cube->molecule.qtot = 2;

	
	// read the grid shape
	cube->gridSize.x = 64;
	cube->gridSize.y = 64;
	cube->gridSize.z = 64;
	cube->npts = cube->gridSize.x * cube->gridSize.y * cube->gridSize.z;
	printf("grid shape: [%i %i %i]\n", cube->gridSize.x, cube->gridSize.y, cube->gridSize.z);

	cube->gpu_grid.x = cube->gridSize.x / 8;
	cube->gpu_grid.y = cube->gridSize.y / 8;
	cube->gpu_grid.z = cube->gridSize.z / 8;
	printf("GPU GRID: [%i %i %i]\n", cube->gpu_grid.x, cube->gpu_grid.y, cube->gpu_grid.z);


	// read the grid origin
	cube->grid0.x = -5.0f * ANG2BOR;
	cube->grid0.y = -5.0f * ANG2BOR;
	cube->grid0.z = -5.0f * ANG2BOR;
	printf("grid origin: %f %f %f \n", cube->grid0.x, cube->grid0.y, cube->grid0.z);

	// translate coordinates so that 0 is at the origin of the grid
	for (int i=0; i<natm; ++i) {
		cube->molecule.coords[i].x -= cube->grid0.x;
		cube->molecule.coords[i].y -= cube->grid0.y;
		cube->molecule.coords[i].z -= cube->grid0.z;
	}

	// read the density cube
	cube->Q = (number*)calloc(sizeof(number), cube->npts);
	printf("cube read.\n");
}


void cube_debug_print(Convolver *cnv, Cube *ref, number *gpusrc, const char *filename) {

	number *dst = (number*)malloc(sizeof(number) * ref->npts);
	cpu_cube_unwrap(cnv, ref, gpusrc, cnv->d_A0n);
	hipMemcpy(dst, cnv->d_A0n, sizeof(number) * ref->npts, hipMemcpyDeviceToHost);
	
	unsigned int gpub = 8;

	FILE *fbin = fopen(filename, "wb");
	fwrite(&ref->npts, sizeof(unsigned int), 1, fbin);
	fwrite(&ref->gridSize, sizeof(dim3), 1, fbin);
	fwrite(&gpub, sizeof(unsigned int), 1, fbin);
	fwrite(dst, sizeof(number), ref->npts, fbin);
	fclose(fbin);


	free(dst);
}














