#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <assert.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "qmtools.h"

#include "basisset.h"
#include "molecule.h"


/*#include "kernel-rpts.h"
#include "kernel-acsf.h"
#include "kernel-qube.h"*/


int nTypes = 10;
int types[] = {1,6,7,8,9,14,15,16,17,35};
__constant__ int c_types[NTYPES];


//Initialization of QMTools, copy types to device memory
void qm_ini(QMTools *obj) {

	printf("ntypes: %i\n", NTYPES);
	for(int i=0; i<NTYPES; i++) {
		printf("%i ", types[i]);
	}
	printf("\n");
	hipMemcpyToSymbol(HIP_SYMBOL(c_types), types, sizeof(float)*NTYPES);
}



void qm_del(QMTools *obj) {

	//scsf_curand_free(obj);
}


//Initialization of Grid variables, copy qube and variables npts and nfields to device memory

void qm_grid_ini(Grid *g) {

	hipError_t hipError_t;
	hipError_t = hipMalloc((void**)&g->d_qube, sizeof(float)*g->npts*g->nfields); assert(hipError_t == hipSuccess);
}

void qm_grid_del(Grid *obj) {

	hipError_t hipError_t;
	hipError_t = hipFree(obj->d_qube); assert(hipError_t == hipSuccess);
}


//Write Grid and Molecule variables to .bin file

void qm_gridmol_write(Grid *g, Molecule *m, const char* filename) {

	FILE *fbin = fopen(filename, "wb");

	fwrite(&m->natoms, sizeof(int), 1, fbin);
	fwrite(m->types, sizeof(int), m->natoms, fbin);
	fwrite(m->coords, sizeof(float3), m->natoms, fbin);

	fwrite(&g->origin, sizeof(float3), 1, fbin);
	fwrite(&g->shape, sizeof(dim3), 1, fbin);
	fwrite(&g->npts, sizeof(uint), 1, fbin);
	fwrite(&g->step, sizeof(float), 1, fbin);
	printf("npts: %i",g->npts);
	fwrite(g->qube, sizeof(float), g->npts, fbin);
	printf("oh yea\n");
	
	fclose(fbin);
}


//???

 SCSFGPU* scsf_gpu_allocate(QMTools *obj, Molecule *m) {

 	SCSFGPU *g = (SCSFGPU*)malloc(sizeof(SCSFGPU));
 	/*
	hipError_t hipError_t;
	int natm = m->natoms;
	int norb = m->norbs;
	int nrpts = obj->nrpts;
	int nacsf = obj->nacsf;

	// constants to constant memory
	int ns[2] = {natm, 0};
	hipMemcpyToSymbol(HIP_SYMBOL(c_esph_ns), ns, sizeof(int)*2);


	// atomic types
	hipError_t = hipMalloc((void**)&g->Zs, sizeof(int)*natm); assert(hipError_t == hipSuccess);
	hipError_t = hipMemcpy(g->Zs, m->types, sizeof(int)*natm, hipMemcpyHostToDevice); assert(hipError_t == hipSuccess);

	// coordinates
	hipError_t = hipMalloc((void**)&g->coords, sizeof(float3)*natm); assert(hipError_t == hipSuccess);
	hipError_t = hipMemcpy(g->coords, m->coords, sizeof(float3)*natm, hipMemcpyHostToDevice); assert(hipError_t == hipSuccess);

	// ALMOs
	hipError_t = hipMalloc((void**)&g->almos, sizeof(short4)*norb); assert(hipError_t == hipSuccess);
	hipError_t = hipMemcpy(g->almos, m->ALMOs, sizeof(short4)*norb, hipMemcpyHostToDevice); assert(hipError_t == hipSuccess);


	// density
	hipError_t = hipMalloc((void**)&g->qube, sizeof(float)*m->npts); assert(hipError_t == hipSuccess);
	hipError_t = hipMalloc((void**)&g->dm, sizeof(float)*norb*norb); assert(hipError_t == hipSuccess);
	hipError_t = hipMemcpy(g->dm, m->dm, sizeof(float)*norb*norb, hipMemcpyHostToDevice); assert(hipError_t == hipSuccess);

	// rpts probability fields
	hipError_t = hipMalloc((void**)&g->VNe000, sizeof(float)*m->npts); assert(hipError_t == hipSuccess);
	hipError_t = hipMalloc((void**)&g->VNe00z, sizeof(float)*m->npts); assert(hipError_t == hipSuccess);
	hipError_t = hipMalloc((void**)&g->VNe0yz, sizeof(float)*m->grid.x*m->grid.y); assert(hipError_t == hipSuccess);
	hipError_t = hipMalloc((void**)&g->VNexyz, sizeof(float)*m->grid.x); assert(hipError_t == hipSuccess);

	// rpts space
	hipError_t = hipMalloc((void**)&g->rpts, sizeof(float3) * nrpts); assert(hipError_t == hipSuccess);

	// acsfs
	hipError_t = hipMalloc((void**)&g->acsf, sizeof(float) * nacsf); assert(hipError_t == hipSuccess);	


	*/
 	return g;
 }

 void scsf_gpu_free(SCSFGPU* g) {

 	hipFree(g->VNe000);
 	hipFree(g->VNe00z);
 	hipFree(g->VNe0yz);
 	hipFree(g->VNexyz);
 	hipFree(g->Zs);
 	hipFree(g->coords);
 	hipFree(g->qube);
 	hipFree(g->rpts);
 	hipFree(g->acsf);

 	free(g);
 }


void scsf_compute(QMTools *obj, Molecule *mol) {


	SCSFGPU *g = scsf_gpu_allocate(obj, mol);


	// compute the random points
	//scsf_getpoints(obj, mol, g);

	// compute acsfs
	//acsf_compute(obj, mol, g);

	// compute density at the points
	//qube(obj, mol, g);

	/*
	random_setup(mol);
	acsf_allocate();


	// compute the positions of evaluation points
	random_grid(mol);
	// compute the cube
	qube(mol);
	// compute acsfs
	acsf_compute(mol);

	//cube_print_unwrap_ongpu(mol, mol->d_VNe, "molecule_29766_vne.bin");
	//cube_print_unwrap_ongpu(mol, mol->d_VNe_az, "molecule_29766_vne_az.bin");
	//cube_print_unwrap2d_ongpu(mol, mol->d_VNe_az_ay, "molecule_29766_vne_az_ay.bin");
	

	// write the complete output

	sprintf(moldir, "molecule_%i_output.bin", cID);
	FILE *fbin = fopen(moldir, "wb");
	
	// write the molecule
	molecule_write_bin(mol, fbin);

	// write the random evaluation points
	fwrite(&nrpts, sizeof(int), 1, fbin);
	fwrite(h_rpts, sizeof(float3), nrpts, fbin);

	// write the acsfs
	fwrite(&nacsf, sizeof(int), 1, fbin);
	fwrite(h_acsf, sizeof(float), nacsf, fbin);

	// write the correct density in those points
	fwrite(h_qube, sizeof(float), nrpts, fbin);

	fclose(fbin);

	molecule_free_complete(mol);

	random_free();
	acsf_free();
	*/


	scsf_gpu_free(g);
}