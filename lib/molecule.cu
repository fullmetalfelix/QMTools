#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "qmtools.h"
#include "basisset.h"
#include "molecule.h"
#include "vectors.h"



/// Computes the grid properties
// coordinates must already be in bohr at this point
// no allocation is done here
void molecule_init(Molecule *mol) {

	int natm = mol->natoms;
	int norbs = mol->norbs;

	hipError_t hipError_t;
	hipError_t = hipMalloc((void**)&mol->d_types, sizeof(int)*natm); assert(hipError_t == hipSuccess);
	hipError_t = hipMalloc((void**)&mol->d_coords, sizeof(float3)*natm); assert(hipError_t == hipSuccess);

	hipError_t = hipMemcpy(mol->d_types, mol->types, sizeof(int)*natm, hipMemcpyHostToDevice); assert(hipError_t == hipSuccess);
	hipError_t = hipMemcpy(mol->d_coords, mol->coords, sizeof(float3)*natm, hipMemcpyHostToDevice); assert(hipError_t == hipSuccess);


	hipError_t = hipMalloc((void**)&mol->d_dm, sizeof(float)*norbs*norbs); assert(hipError_t == hipSuccess);
	hipError_t = hipMemcpy(mol->d_dm, mol->dm, sizeof(float)*norbs*norbs, hipMemcpyHostToDevice); assert(hipError_t == hipSuccess);

	hipError_t = hipMalloc((void**)&mol->d_ALMOs, sizeof(short4)*norbs); assert(hipError_t == hipSuccess);
	hipError_t = hipMemcpy(mol->d_ALMOs, mol->ALMOs, sizeof(short4)*norbs, hipMemcpyHostToDevice); assert(hipError_t == hipSuccess);
}



void molecule_del(Molecule *m) {

	hipError_t hipError_t;
	hipError_t = hipFree(m->d_types); assert(hipError_t == hipSuccess);
	hipError_t = hipFree(m->d_coords); assert(hipError_t == hipSuccess);
	hipError_t = hipFree(m->d_dm); assert(hipError_t == hipSuccess);
	hipError_t = hipFree(m->d_ALMOs); assert(hipError_t == hipSuccess);
}


// NOT USED!
// this function has to create a Grid from scratch and return it
void molecule_densitygrid(Molecule *m, Grid *g, float step, float fat) {


	float3 crdmax = make_float3(-FLT_MAX,-FLT_MAX,-FLT_MAX);
	float3 crdmin = make_float3( FLT_MAX, FLT_MAX, FLT_MAX);

	for(int i=0; i<m->natoms; i++) {
		
		crdmax = float3_max(crdmax, m->coords[i]);
		crdmin = float3_min(crdmin, m->coords[i]);
	}

	crdmin = crdmin -fat * ANG2BOR;
	crdmax = crdmax +fat * ANG2BOR;
	
	float3 grd = crdmax-crdmin;
	g->origin = crdmin;

	//printf("grid real size: %f %f %f\n", grd.x, grd.y, grd.z);
	//printf("grid origin: %f %f %f\n", crdmin.x, crdmin.y, crdmin.z);

	grd = grd / step;
	grd = grd / 8.0f;
	grd = float3_ceiled(grd);
	grd = grd * 8;
	
	g->shape = dim3(grd.x, grd.y, grd.z);
	g->GPUblocks = dim3(g->shape.x / 8, g->shape.y / 8, g->shape.z / 8);
	g->npts = g->shape.x * g->shape.y * g->shape.z;


	//printf("grid shape: %i %i %i - total: %i\n", mol->grid.x,mol->grid.y,mol->grid.z,mol->npts);
	//printf("grid blocks: %i %i %i\n", mol->blocks.x,mol->blocks.y,mol->blocks.z);

}



/*

void molecule_gpu_init(Molecule *mol) {

	hipError_t hipError_t;
	int natm = mol->natoms;

	// atomic types
	int *d_Zs;
	hipError_t = hipMalloc((void**)&d_Zs, sizeof(int)*natm); assert(hipError_t == hipSuccess);
	hipMemcpy(d_Zs, mol->types, sizeof(int)*natm, hipMemcpyHostToDevice); assert(hipError_t == hipSuccess);
	mol->d_Zs = d_Zs;

	// coordinates
	float3 *d_coords, *coords;
	coords = (float3*)malloc(sizeof(float3)*natm);
	memcpy(coords, mol->coords, sizeof(float3)*natm);
	for(int i=0; i<mol->natoms; i++) // move every atom so that grid0 is 0!
		coords[i] = coords[i] - mol->grid0;
	hipError_t = hipMalloc((void**)&d_coords, sizeof(float3)*natm); assert(hipError_t == hipSuccess);
	hipMemcpy(d_coords, coords, sizeof(float3)*natm, hipMemcpyHostToDevice); assert(hipError_t == hipSuccess);
	memcpy(mol->coords, coords, sizeof(float3) * natm);
	mol->d_coords = d_coords;
	free(coords);

	hipError_t = hipMalloc((void**)&mol->d_refQ, sizeof(float)*mol->npts); assert(hipError_t == hipSuccess);
	hipError_t = hipMalloc((void**)&mol->d_nnQ,  sizeof(float)*mol->npts); assert(hipError_t == hipSuccess);
	hipError_t = hipMalloc((void**)&mol->d_VNe,  sizeof(float)*mol->npts); assert(hipError_t == hipSuccess);
	hipError_t = hipMalloc((void**)&mol->d_VNe_az, sizeof(float)*mol->npts); assert(hipError_t == hipSuccess);
	hipError_t = hipMalloc((void**)&mol->d_VNe_az_ay, sizeof(float)*mol->grid.x*mol->grid.y); assert(hipError_t == hipSuccess);
	hipError_t = hipMalloc((void**)&mol->d_VNe_az_ay_ax, sizeof(float)*mol->grid.x); assert(hipError_t == hipSuccess);
}


void molecule_gpu_free(Molecule *mol) {
	hipFree(mol->d_Zs);
	hipFree(mol->d_coords);
	hipFree(mol->d_refQ);
	hipFree(mol->d_nnQ);
	hipFree(mol->d_VNe);
	hipFree(mol->d_VNe_az);
	hipFree(mol->d_VNe_az_ay);
	hipFree(mol->d_VNe_az_ay_ax);
}



void molecule_write_bin(Molecule *mol, FILE *fbin) {

	fwrite(&mol->natoms, sizeof(int), 1, fbin);
	for(int i=0; i<mol->natoms; i++) {
		fwrite(&mol->types[i], sizeof(int), 1, fbin);
		fwrite(&mol->coords[i], sizeof(float3), 1, fbin);
	}
}
*/